//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2 //1024

__global__ void definitions (int* A, unsigned int* B, unsigned long long int* C)
{
  atomicCAS(A,2,11);

  atomicCAS(B,5,1);

  atomicCAS(C,7,3);
}
